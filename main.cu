/**
 * A C application that uses CUDA to generate a Mandelbrot image.
 * Designed to work on my NVIDIA GeForce GTX 1650
 */


#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdint>
#include <cstdlib>
#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>

// a structure to represent a complex number
typedef struct _complex {
    float re;
    float im;
} Complex;

// find the complex magnitude
// note: we're not taking the square-root here
__device__ 
float complex_magnitude(Complex* c) {
    return (c->re * c->re) + (c->im * c->im);
}

// add two complex numbers into a given result complex number
__device__ 
void complex_add(Complex* z, Complex* other, Complex* result) {
    result->re = z->re + other->re;
    result->im = z->im + other->im;
}

// multiply two complex numbers into a given result complex number
__device__ 
void complex_multiply(Complex* x, Complex* y, Complex* result) {
    result->re = (x->re * y->re) - (x->im * y->im);
    result->im = (x->re * y->im) + (x->im * y->re);
}

__device__
void get_color(int t, uint8_t* r, uint8_t* g, uint8_t* b) {
    uint8_t palette[][3] = {
        {66, 30, 15},
        {25, 7, 26},
        {9, 1, 47},
        {4, 4, 73},
        {0, 7, 100},
        {12, 44, 138},
        {24, 82, 177},
        {57, 125, 209},
        {134, 181, 229},
        {211, 236, 248},
        {241, 233, 191},
        {248, 201, 95},
        {255, 170, 0},
        {204, 128, 0},
        {153, 87, 0},
        {106, 52, 3},
    };

    int i = t % 16;
    *r = palette[i][0];
    *g = palette[i][1];
    *b = palette[i][2];
}

__global__
void mandelbrot(uint8_t* device_image, int width, int height, int max_iters) {
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;

    int index = (y * width + x) * 3;

    if (x < width && y < height) {
        uint8_t r;
        uint8_t g;
        uint8_t b;

        float u = (float)x / height;
        float v = (float)y / height;

        Complex z = {0.0f, 0.0f};
        Complex c = {2.5f * (u - 0.5f) - 1.4f, 2.5f * (v - 0.5f)};

        int i = 0;
        while (i < max_iters && complex_magnitude(&z) < 32.0f) {
            Complex zsq;
            complex_multiply(&z, &z, &zsq);
            complex_add(&zsq, &c, &z);
            i += 1;
        }
        
        float t = (float)i - logf(logf(complex_magnitude(&z)));

        get_color((int)t, &r, &g, &b);

        device_image[index] = r; 
        device_image[index + 1] = g;
        device_image[index + 2] = b; 
    }
}


int main() {
    const int width = 1920;
    const int height = 1080;
    const int max_iterations = 255;

    uint8_t* host_img = (uint8_t*)malloc(width * height * 3);
    if (!host_img) {
        fprintf(stderr, "could not malloc the host image!\n");
        return -1;
    }

    uint8_t* device_img;
    hipMalloc(&device_img, width * height * 3);
    
    dim3 blocks(32, 32); // max 1024 threads per block on my setup
    dim3 grid(ceil((float)width / 32), ceil((float)height / 32));

    mandelbrot<<<grid, blocks>>>(device_img, width, height, max_iterations);
    hipDeviceSynchronize();

    hipMemcpy(host_img, device_img, width * height * 3, hipMemcpyDeviceToHost);
    
    // write an output image
    FILE* outfile = fopen("/home/pimeson/temp/mandelbrot_cuda.ppm", "w+");
    fprintf(outfile, "P6\n%d %d\n255\n", width, height);
    fwrite(host_img, width * height * 3, sizeof(uint8_t), outfile); 
    fclose(outfile);
    
    free(host_img);
    hipFree(device_img);

    printf("wrote an image\n");

    return 0;
}

/*
```
❯ nvcc -o main main.cu

// for debug using cuda-gdb
❯ nvcc -g -G -o main main.cu
```
*/

